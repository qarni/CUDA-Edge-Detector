#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>

#include <cutil.h>
#include "canny.h"

// kernel function definitions

// device function defintions 
__device__ int getPixelVal(int* image, int height, int width, int x, int y);
__global__ void GaussianBlur(int* input, int* output, int height, int width, float* gaussianFilter, int kernelSize, int32_t* count);
__global__ void FindGradients(int* input, int* output, int height, int width);

// ------------------------------------------------------------------------------------

/*
Wrapper function to make kernel calls to perform canny algorithm 
*/
void canny(int* input, int height, int width, int* output, int kernelSize,  int sigma) {

    clock_t before = clock();

    int matrixSize = height * width * sizeof(int);
    float* filter  = generateGaussianFilter(kernelSize, sigma);

    // set up for kernel calls 
    float* filterD;
    int32_t count = 0;
    int32_t* countD;

    int* inputD = AllocateDeviceMemory(matrixSize);
    int* gaussianBlurD = AllocateDeviceMemory(matrixSize);
    int* outputD = AllocateDeviceMemory(matrixSize);
    hipMalloc(&filterD, kernelSize * kernelSize * sizeof(float));
    hipMalloc(&countD, sizeof(int32_t));
    
    CopyToDevice(&(input[0]), inputD, matrixSize);
    CopyToDevice(&(output[0]), outputD, matrixSize);
    CopyToDevice(&(filter[0]), filterD, kernelSize * kernelSize * sizeof(float));
    CopyToDevice(&count, countD, sizeof(int32_t));

    // set up dimensions for calls to kernel -------------------------------------------------------------------

    // 2400 = 8  * 300
    // 600  = 8 * 75
    // 4 = 1  * 4

    dim3 threadsPerBlock(8, 8);
    dim3 numBlocks(300, 300);

    GaussianBlur<<<numBlocks,threadsPerBlock>>>(inputD, gaussianBlurD, height, width, filterD, kernelSize, countD);
    hipDeviceSynchronize();

    FindGradients<<<numBlocks, threadsPerBlock>>>(gaussianBlurD, outputD, height, width);
    hipDeviceSynchronize();

    // tear down after kernel calls are done -------------------------------------------------------------------
    CopyFromDevice(outputD, &(output[0]), matrixSize);
    CopyFromDevice(countD, &count, sizeof(int32_t));
    hipFree(inputD);
    hipFree(outputD);
    hipFree(filterD);
    hipFree(countD);

    printf("\n\ndone with canny algorithm\n");
    printf("count: %d\n", count);

    clock_t difference = clock() - before;
    int msec = difference * 1000 / CLOCKS_PER_SEC;
    printf("Time taken %d seconds %d milliseconds\n", msec/1000, msec%1000);

}

/*
This is the guassian filter to be applied over each pixel in the image


G(x, y) = (1/2*pi*sigma^2)*e^-(x^2+y^2/2*sigma^2)
*/
float*  generateGaussianFilter(int kernelSize, int sigma) {

    float* filter  = (float*) malloc(kernelSize * kernelSize * sizeof(float));

    float div = 2.0 * sigma *  sigma;
    float pre = 1.0 / (M_PI * div);

    int i  = 0;
    
    for (int x = -2; x <= 2; x++) { 
        for (int y = -2; y <= 2; y++) { 
            filter[i] = pre * pow(M_E, -((pow(x,2) + pow(y, 2)) / div));
            i++;
        }
    }

    return filter;
}

// kernel functions --------------------------------------------------------------------

/*
Noise reduction - gets rid of background noise but still keeps borders more in focus so they 
can be detected in the next step

Apply gaussian filter over each pixel 
Start with kernel size of 5?

For the borders of the image (anything less than the size of the kernel away from an edge)
Just use the original vals of the image

*/
__global__ void GaussianBlur(int* input, int* output, int height, int width, float* gaussianFilter, int kernelSize, int32_t* count) {

    int row = (blockIdx.x * blockDim.x) + threadIdx.x;
    int col = (blockIdx.y * blockDim.y) + threadIdx.y;

    output[width * row + col] = -5;

    int val = getPixelVal(input, height, width, row, col);
    if(val == -1){
        return;
    }
        
    int kernelHalf = kernelSize/2;

    // account for borders of the image which can't have the filter applied to them
    if(row < kernelHalf || col < kernelHalf || row > width - 1 - kernelHalf || col > height - 1 - kernelHalf) {
        output[width * row + col] = val;
    }
    // otherwise, apply the filter!
    else {

        float filteredVal = 0.0;
        int f = 0;
        for(int krow = -kernelHalf; krow <= kernelHalf; krow++) {
            for(int kcol = -kernelHalf; kcol <= kernelHalf; kcol++) {
                filteredVal += (float)getPixelVal(input, height, width, row + krow, col + kcol) * gaussianFilter[f];
                f++;
            }
        }
        
        output[width * row + col] = (int)filteredVal;
    }

    __syncthreads();

    atomicAdd(count, 1);
}

/*
Find gradients - this is the step that actually detects edges (roughly)

Very similar to previous step, just need to apply Sobel filters this time

Kx = -1 0 1 -2 0 2 -1 0 1
Ky = 1 2 1 0 0 0 -1 -2 -1

Also need this data for later:

Magnitude G = sqrt(Ix^2 + Iy^2)
slope O grad = arctan(Iy/Ix)

*/
__global__ void FindGradients(int* input, int* output, int height, int width) {

    // sobel filters. Apply both!
    int Kx[] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
    int Ky[] = {1, 2, 1, 0, 0, 0, -1, -2, -1};

    int row = (blockIdx.x * blockDim.x) + threadIdx.x;
    int col = (blockIdx.y * blockDim.y) + threadIdx.y;

    int val = getPixelVal(input, height, width, row, col);
    if(val == -1){
        return;
    }

    // account for borders of the image which can't have the filter applied to them
    if(row < 1 || col < 1 || row > width - 2 || col > height - 2) {
        output[width * row + col] = val;
    }
    // otherwise, apply the filters!
    else {

        float filteredValX = 0.0;
        float filteredValY = 0.0;
        int f = 0;
        for(int krow = -1; krow <= 1; krow++) {
            for(int kcol = -1; kcol <= 1; kcol++) {
                filteredValX += (float)getPixelVal(input, height, width, row + krow, col + kcol) * Kx[f];
                filteredValY += (float)getPixelVal(input, height, width, row + krow, col + kcol) * Ky[f];
                f++;
            }
        }

        float sobel = sqrt(pow(filteredValX, 2) + pow(filteredValY, 2));
        
        output[width * row + col] = (int)sobel;

        // TODO: calc gradient direction
    }

    __syncthreads();

}

// device functions --------------------------------------------------------------------
// can only be called from global func or from another device func, not from host

/*
returns pixel value at a location 
Maps a 2d image to a 1d list

if error, returns -1

origAddress + (width * row + col)
*/
__device__ int getPixelVal(int* image, int height, int width, int row, int col) {
    if (col < height && row < width && col >= 0 && row >= 0)
        return *(image + width * row + col);
    else{
        printf("CRAP");
        return -1;
    }
}

// helper functions -------------------------------------------------------------------

int* AllocateDeviceMemory (int size){
    int* res;
    hipMalloc(&res, size);
    return res;
}

void CopyFromDevice(void* mDevice, void* mHost, int size){
    hipMemcpy(mHost, mDevice, size, hipMemcpyDeviceToHost);
}

void CopyToDevice(void* mHost,  void* mDevice, int size){
    hipMemcpy(mDevice, mHost, size, hipMemcpyHostToDevice);
}

